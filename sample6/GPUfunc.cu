
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
using namespace std;

namespace GPUfunc{
	static int nb    = 1024; //1024*1024*64*2; // max 1024*1024*64*2
	static int nthre = 1; // max 65535
	static int nthre_total = nb*nthre;
	static int nword =  1024*1024*8;
	static int mem_size = sizeof(double) * nword;
	static int mem_size_o = nthre_total*sizeof(double);
	static double* hmem_i;
	static double* hmem_o;
	static double* dmem_i;
	static double* dmem_o;
	static int mpi_proc_id;

#define NLOOP (1000)

	__device__ double myDeviceFunc_0(double* in, int nword)
	{
		double z=0.;
		while(z < 7.777777){
				z += 1.0e-5;
		}
		return ((double)z);
	} 


	__global__ void kernel(double* in, double* out, int nword)
	{
		int tid = threadIdx.x;
		int bid = blockIdx.x;
		int global_id = blockDim.x*bid + tid;
		double z;

		z = myDeviceFunc_0(in, nword);

		out[global_id] = z;
	}

	void initialize(int _mpi_proc_id)
	{
		static bool is_first = true;
		if(false == is_first) return;

		mpi_proc_id = _mpi_proc_id;

		int GPU_N;
		hipGetDeviceCount(&GPU_N);
		///		if(0 == mpi_proc_id) printf("CUDA-capable device count: %i\n", GPU_N);

		int devid = mpi_proc_id % GPU_N;
		hipSetDevice(devid); // <--------------------------- Select a GPU
		printf("[%d] using GPU[%d]\n", mpi_proc_id, devid);

		// input buffer (Host)
		hmem_i = (double*) malloc(mem_size);
		for(int i=0; i<nword; i++) hmem_i[i] = (double)i;
		// input buffer (GPU)
		hipMalloc( (void**) &dmem_i, mem_size);
		hipMemcpy(dmem_i, hmem_i, mem_size, hipMemcpyHostToDevice);
		// output buffer (Host/GPU)
		hipMalloc( (void**) &dmem_o, mem_size_o);
		hmem_o = (double*) malloc(mem_size_o);

		is_first = false;
	}

	void run()
	{
		kernel<<< nb, nthre>>>(dmem_i, dmem_o, nword);
		hipMemcpy(hmem_o, dmem_o, mem_size_o, hipMemcpyDeviceToHost);

		//		printf("[%d] %d, %e\n", mpi_proc_id, nthre_total-1, hmem_o[nthre_total-1]);

		return;
	}

	void finalize(){
		free(hmem_i);
		free(hmem_o);
		hipFree(dmem_i);
		hipFree(dmem_o);
	}

}

