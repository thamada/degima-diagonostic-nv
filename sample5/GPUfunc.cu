
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
using namespace std;

namespace GPUfunc{
	static int nb    = 64; //1024*1024*64*2; // max 1024*1024*64*2
	static int nthre = 1; // max 65535
	static int nthre_total = nb*nthre;
	static int nword =  1024*1024*8;
	static int mem_size = sizeof(double) * nword;
	static int mem_size_o = nthre_total*sizeof(double);
	static double* hmem_i;
	static double* hmem_o;
	static double* dmem_i;
	static double* dmem_o;

#define NLOOP (1000)
#define NX (14705)

	__device__ double myDeviceFunc_0(double* in, int nword)
	{
		double z=0.0;
		double x[NX];
		for(int i=0; i<NX; i++)  x[i] = 1.0;
		for(int j=0; j<NLOOP; j++)	for(int i=0; i<NX; i++)  z += x[i];
		return (z);
	} 

	__device__ double myDeviceFunc_1(double* in, int nword)
	{
		double z=0.0;
		double x[NX];
		for(int i=0; i<NX; i++)  x[i] = 1.0;
		for(int j=0; j<NLOOP; j++)	for(int i=0; i<NX; i++)  z += x[i];
		return (z);
	} 

	__device__ double myDeviceFunc_2(double* in, int nword)
	{
		double z=0.0;
		double x[NX];
		for(int i=0; i<NX; i++)  x[i] = 1.0;
		for(int j=0; j<NLOOP; j++)	for(int i=0; i<NX; i++)  z += x[i];
		return (z);
	} 

	__device__ double myDeviceFunc_3(double* in, int nword)
	{
		double z=0.0;
		double x[NX];
		for(int i=0; i<NX; i++)  x[i] = 1.0;
		for(int j=0; j<NLOOP; j++)	for(int i=0; i<NX; i++)  z += x[i];
		return (z);
	} 

	__global__ void kernel(double* in, double* out, int nword)
	{
		int tid = threadIdx.x;
		int bid = blockIdx.x;
		int global_id = blockDim.x*bid + tid;
		double z;

		int kernel_num = global_id % 4;

		switch(kernel_num){
		case 0: 
			z = myDeviceFunc_0(in, nword);
			break;
		case 1: 
			z = myDeviceFunc_1(in, nword);
			break;
		case 2: 
			z = myDeviceFunc_2(in, nword);
			break;
		case 3: 
			z = myDeviceFunc_3(in, nword);
			break;
		default:
			z = myDeviceFunc_0(in, nword);
		}
		out[global_id] = z;
	}

	void initialize()
	{
		static bool is_first = true;
		if(false == is_first) return;

		// input buffer (Host)
		hmem_i = (double*) malloc(mem_size);
		for(int i=0; i<nword; i++) hmem_i[i] = (double)i;
		// input buffer (GPU)
		hipMalloc( (void**) &dmem_i, mem_size);
		hipMemcpy(dmem_i, hmem_i, mem_size, hipMemcpyHostToDevice);
		// output buffer (Host/GPU)
		hipMalloc( (void**) &dmem_o, mem_size_o);
		hmem_o = (double*) malloc(mem_size_o);

		printf("# threads:   %d \n", nthre_total);
		printf("mem_size:    %d MB\n", mem_size >> 20);
		printf("mem_size_o:    %d kB\n", mem_size_o >> 10);
		is_first = false;
	}

	void run()
	{
		kernel<<< nb, nthre>>>(dmem_i, dmem_o, nword);
		hipMemcpy(hmem_o, dmem_o, mem_size_o, hipMemcpyDeviceToHost);

		/*
		for(int i=0; i<nthre_total; i++){
			double z = hmem_o[i];
			if(i>(nthre_total-4)) printf("%d, %f\n", i, z);
		}
		*/

		printf("%d, %e\n", nthre_total-1, hmem_o[nthre_total-1]);

		return;
	}

	void finalize(){
		free(hmem_i);
		free(hmem_o);
		hipFree(dmem_i);
		hipFree(dmem_o);
	}

}

