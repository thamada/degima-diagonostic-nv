#include "hip/hip_runtime.h"
//Time-stamp: <2013-12-04 13:20:59 hamada>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>
//#define NUM (1024*1024*1024/4) // 1GB
//#define NUM ((1024*1024/4)*(1024 + 512)) // 1.5GB
//#define NUM ((1024*1024/4)*(1024*2) -1) // 2GB-1
//#define NUM ((1024*1024/4)*(1024*2) ) // 2GB
//#define NUM ((1024*1024/4)) // 1MB
#define NUM ((1024*1024/4)*(1024*6-100) ) // 6GB-100MB
using namespace std;

__global__ void kernel(int* x, int n)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
	int wid = blockDim.x*bid + tid;
	if(wid > 1) return;
	int z = 0;
	for(size_t i = 0; i<n ; i++)		z += x[i];
	x[0] = z;
}

#include <sys/time.h>
#include <sys/resource.h>
extern "C" double get_time(void)
{
  static struct timeval tv;
  static struct timezone tz;
  gettimeofday(&tv, &tz);
  return ((double)(tv.tv_sec  + tv.tv_usec*1.0e-6));
}

void myCudaMalloc(void** val, size_t mem_size)
{
	double t = get_time();
	hipError_t err = hipMalloc(val, mem_size);
	assert(hipSuccess == err);
	cout << "hipMalloc: " << get_time() - t << endl; 
}

void myCudaMemcpy(void* dst, const void* src, size_t size, enum hipMemcpyKind kind)
{
	double t = get_time();
	hipError_t err = hipMemcpy(dst, src, size, kind);
	assert(hipSuccess == err);
	cout << "hipMemcpy: " << get_time() - t << endl; 
}

int main( int argc, char** argv) 
{
	int nb    = 512; // max 65535
	int nthre = 128; // max 512
	size_t nword = NUM;
	size_t mem_size = sizeof(int) * nword;
	printf("# nword:     %zd \n", nword);
	printf("# threads:   %d  \n", nb*nthre);
	printf("mem_size:    %zd Kbyte\n", mem_size >> 10);
	double t=0.;
	hipError_t err;
	int* hval = (int*) malloc(mem_size);
	int* hval2 = (int*) malloc(mem_size);
	int* dval = NULL;
	cout << "mem_size:        " <<mem_size << endl;
	cout << "(size_t)mem_size:" <<(size_t)mem_size << endl;
	cout << sizeof(size_t) << endl;

	myCudaMalloc((void**)&dval,  mem_size);

	int z = 0;
	for(size_t i=0; i<nword; i++){hval[i]  = 1;  z += hval[i];}

	myCudaMemcpy(dval,  hval,  mem_size, hipMemcpyHostToDevice);

	t = get_time();
	kernel<<< nb, nthre >>>(dval, nword);
	err = hipDeviceSynchronize();
	assert(hipSuccess == err);
	cout << "GPU calc:   " << get_time() - t << endl; 

	myCudaMemcpy(hval,  dval,  mem_size, hipMemcpyDeviceToHost);

	printf("GPU: %d\n", hval[0]);
	printf("HOS: %d\n", z);

	free(hval);
	hipFree(dval);


	return (0);
}
