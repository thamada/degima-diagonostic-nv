
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <math.h>
//#include <cutil.h>

// �ǥХ����ؿ�(GPU¦�Ǽ¹Ԥ�������򵭽�)
// �������
// GPU���꤫��ǡ������äƤ��� +1 ����GPU������᤹
// �Ȥ��������Ǥ�
__global__ void function_on_GPU(float* d_idata, float* d_odata, int nword)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  if((tid == 0) && (bid==0)){      // ��ñ�Τ��������GPU���1�ĤΥ���åɤ˸���
    for(int i = 0; i<nword; i++){
      d_odata[i] =  d_idata[i] + 1.0f ;
    }
  }
}


// main�ؿ�.
// ���ʤߤ� __global__���դ��Ƥ��ʤ��ؿ��������̾��C++�����ɤȤ���CPU¦�Ǽ¹Ԥ���ޤ�.
// (�Ĥޤ� __global__�ʴؿ����ʤ����ñ�ʤ�C++�����ɤʤΤ�g++�ǥ���ѥ����ǽ�Ǥ�)

int main( int argc, char** argv) 
{
    int nword = 1024;
    int mem_size = sizeof(float) * nword;

    // �ۥ��ȥ���(CPU¦)����
    float* h_idata = (float*) malloc(mem_size);
    float* h_odata = (float*) malloc(mem_size);
    for(unsigned int i = 0; i < nword; ++i){
      h_idata[i] = (float) i;
    }

    // �ǥХ�������(GPU¦)����
    float* d_idata;
    hipMalloc((void**) &d_idata, mem_size);
    float* d_odata;
    hipMalloc( (void**) &d_odata, mem_size);

    // �ǡ���ž��:  �ۥ��ȥ��� -----> �ǥХ�������
    hipMemcpy( d_idata, h_idata, mem_size, hipMemcpyHostToDevice );

    //�ǥХ����ؿ�(GPU��Ǽ¹Ԥ����ؿ�)��¹�
    dim3  grid(128);     // �֥�å���( ���Ѥ���SIMD���å׿� )
    dim3  threads(128);  // ����åɿ�( ���Ѥ��륹��åɿ�(SIMD���å�������))
    // grid*thread(= 128*128)����åɤ�����˥ǥХ����ؿ���¹Ԥ��ޤ�.
    function_on_GPU<<< grid, threads >>>(d_idata, d_odata, nword);

    // �ǡ���ž��:  �ǥХ������� -----> �ۥ��ȥ���
    hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost);

    for(int i=0; i<17; i++){
      printf("%f, %f\n", h_idata[i], h_odata[i]);
    }

    // �����ΰ����
    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);

    return (0);
}


