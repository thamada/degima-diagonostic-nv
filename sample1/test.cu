
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <math.h>
//#include <cutil.h>

// デバイス関数(GPU側で実行する処理を記述)
// この例は
// GPUメモリからデータを取ってきて +1 してGPUメモリに戻す
// という処理です
__global__ void function_on_GPU(float* d_idata, float* d_odata, int nword)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  if((tid == 0) && (bid==0)){      // 簡単のため処理はGPU上の1個のスレッドに限定
    for(int i = 0; i<nword; i++){
      d_odata[i] =  d_idata[i] + 1.0f ;
    }
  }
}


// main関数.
// ちなみに __global__が付いていない関数は全て通常のC++コードとしてCPU側で実行されます.
// (つまり __global__な関数がなければ単なるC++コードなのでg++でコンパイル可能です)

int main( int argc, char** argv) 
{
    int nword = 1024;
    int mem_size = sizeof(float) * nword;

    // ホストメモリ(CPU側)設定
    float* h_idata = (float*) malloc(mem_size);
    float* h_odata = (float*) malloc(mem_size);
    for(unsigned int i = 0; i < nword; ++i){
      h_idata[i] = (float) i;
    }

    // デバイスメモリ(GPU側)設定
    float* d_idata;
    hipMalloc((void**) &d_idata, mem_size);
    float* d_odata;
    hipMalloc( (void**) &d_odata, mem_size);

    // データ転送:  ホストメモリ -----> デバイスメモリ
    hipMemcpy( d_idata, h_idata, mem_size, hipMemcpyHostToDevice );

    //デバイス関数(GPU上で実行される関数)を実行
    dim3  grid(128);     // ブロック数( 使用するSIMDチップ数 )
    dim3  threads(128);  // スレッド数( 使用するスレッド数(SIMDチップ当たり))
    // grid*thread(= 128*128)スレッドが並列にデバイス関数を実行します.
    function_on_GPU<<< grid, threads >>>(d_idata, d_odata, nword);

    // データ転送:  デバイスメモリ -----> ホストメモリ
    hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost);

    for(int i=0; i<17; i++){
      printf("%f, %f\n", h_idata[i], h_odata[i]);
    }

    // メモリ領域解放
    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);

    return (0);
}


