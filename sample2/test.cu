
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
using namespace std;

__global__ void kernel(int* dval, int nword)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
	int i = blockDim.x*bid + tid;
	dval[i] = i;
}

int main( int argc, char** argv) 
{
	/*
	int nb    = 65535; // max 65535
	int nthre = 512; // max 512
	*/
	int nb    = 512; // max 65535
	int nthre = 128; // max 512

	int nword = nb * nthre;
	int mem_size = sizeof(int) * nword;
	printf("# threads:   %d \n", nb*nthre);
	printf("mem_size:    %d Kbyte\n", mem_size >> 10);

	int* hval = (int*) malloc(mem_size);
	int* dval;
	hipMalloc( (void**) &dval, mem_size);

	dim3  grid(nb);
	dim3  threads(nthre);
	kernel<<< grid, threads >>>(dval, nword);

	hipMemcpy(hval, dval, mem_size, hipMemcpyDeviceToHost);

	for(int i=0; i<nword; i++){
		int z = hval[i];
		if(i != z){
			printf("%d, %d\n", i, z);
		}
	}

	free(hval);
	hipFree(dval);
	return (0);
}
