
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
using namespace std;

void calc_on_cpu(float* vec_X, float* vec_Y, float* vec_Z, int nword)
{
	for(int i=0; i<nword; i++){
		vec_Z[i] = vec_X[i] + vec_Y[i];
	}
}


__global__ void kernel(float* vec_X, float* vec_Y, float* vec_Z, int nword)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
	int i = blockDim.x*bid + tid;
	vec_Z[i] = vec_X[i] + vec_Y[i];
}

int main( int argc, char** argv) 
{
	int nb    = 512; // max 65535
	int nthre = 128; // max 512

	int nword = nb * nthre;
	int mem_size = sizeof(float) * nword;
	printf("# threads:   %d \n", nb*nthre);
	printf("mem_size:    %d Kbyte\n", mem_size >> 10);

	float* hval_X = (float*) malloc(mem_size);
	float* hval_Y = (float*) malloc(mem_size);
	float* hval_Z = (float*) malloc(mem_size);
	float* dval_X;
	float* dval_Y;
	float* dval_Z;
	hipMalloc( (void**) &dval_X, mem_size);
	hipMalloc( (void**) &dval_Y, mem_size);
	hipMalloc( (void**) &dval_Z, mem_size);

	for(int i=0; i<nword; i++){
		float a = (float) i;
		hval_X[i] =  a;
		hval_Y[i] = -a;
	}

	hipMemcpy(dval_X, hval_X, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(dval_Y, hval_Y, mem_size, hipMemcpyHostToDevice);

	dim3  grid(nb);
	dim3  threads(nthre);
	kernel<<< grid, threads >>>(dval_X, dval_Y, dval_Z, nword);
	hipMemcpy(hval_Z, dval_Z, mem_size, hipMemcpyDeviceToHost);

	//		calc_on_cpu(hval_X, hval_Y, hval_Z, nword);

	for(int i=0; i<nword; i++){
		printf("%d: %f + %f => %f\n", i, hval_X[i], hval_Y[i], hval_Z[i]);
	}

	free(hval_X);
	free(hval_Y);
	free(hval_Z);
	hipFree(dval_X);
	hipFree(dval_Y);
	hipFree(dval_Z);
	return (0);
}
