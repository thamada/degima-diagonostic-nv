
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <cassert>


__global__ void kernel(int* dval, int nword)
{
  int tid = threadIdx.x;
	int3 bid;
	bid.x = blockIdx.x;
	bid.y = blockIdx.y;

	int nthre = blockDim.x;
	int i   = (gridDim.x*bid.y + bid.x)*nthre + tid;

	dval[i] = i;
}

int main( int argc, char** argv) 
{
	int nby   = 6;
	int nbx   = 65535; // max 65535 blocks

	int nthre = 512;   // max 512 threads
	int nword = nbx * nby * nthre;
	int mem_size = sizeof(int) * nword;
	printf("# threads:   %d \n", nword);
	printf("mem_size:    %d Kbyte\n", mem_size >> 10);

	int* hval = (int*) malloc(mem_size);
	int* dval;
	hipMalloc( (void**) &dval, mem_size);

	dim3  grid(nbx, nby);
	dim3  threads(nthre);
	kernel<<< grid, threads >>>(dval, nword);

	hipMemcpy(hval, dval, mem_size, hipMemcpyDeviceToHost);

	for(int i=0; i<nword; i++){
		int z = hval[i];
		if(i != z) printf("%d: %d\n", i, z);
	}

	free(hval);
	hipFree(dval);
	return (0);
}
