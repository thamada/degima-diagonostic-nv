#include "hip/hip_runtime.h"
//Time-stamp: <2011-01-15 08:20:37 hamada>
//#define DEBUG

#include <iostream>
#include <omp.h>
#include <string.h>
#include <cassert>
#include <cutil.h>

//--------------------------------
#include <sys/time.h>
#include <sys/resource.h>

class Time{
	double t;
public:
	Time() {t = 0.0;}
	struct timeval tv;
	struct timezone tz;
	double get_time(void){
		gettimeofday(&tv, &tz);
		t = (double)(tv.tv_sec  + tv.tv_usec*1.0e-6);
		return (t);
	}
};

__global__ void kernel(int* dval, int nword)
{
  int tid = threadIdx.x;
	int3 bid;
	bid.x = blockIdx.x;
	bid.y = blockIdx.y;
	int nthre = blockDim.x;
	int i   = (gridDim.x*bid.y + bid.x)*nthre + tid;
	dval[i] = ~dval[i];
}

#include <ctime>

void setup_vector(int n, int *x0,  int *x1)
{
	//	srand48(0x19740526);
	srand48((long int)time(NULL));
	for(int i=0; i<n; i++){
		x0[i]= (int)(0xFFFFFFFFull & mrand48());
		x1[i]=x0[i];
	}
}

#include <cutil_inline.h>

class GPU{
public :
	GPU () { }

	void open(int gpuid){
		cutilSafeCall( hipSetDevice(gpuid) );	
	}

	int GetDevice(){
		int x=999;
		cutilSafeCall( hipGetDevice(&x) );
		return x;
	}

	void runtest( int gpuid, int nloop)
	{
		int nby   = 6;
		int nbx   = 65535; // max 65535 blocks
		int nthre = 512;   // max 512 threads
		int nword = nbx * nby * nthre;
		int mem_size = sizeof(int) * nword;

#if defined(DEBUG)
		printf("# threads:   %d \n", nword);
		printf("mem_size:    %d Kbyte\n", mem_size >> 10);
#endif

		int* hval = (int*) malloc(mem_size);
		int* zval = (int*) malloc(mem_size);
		int* dval;
		cutilSafeCall( hipMalloc( (void**) &dval, mem_size) );

		setup_vector(nword, hval, zval);

		Time t;
		double tt;
		tt = t.get_time();

		cutilSafeCall( hipMemcpy(dval, hval, mem_size, hipMemcpyHostToDevice) );
		dim3  grid(nbx, nby);
		dim3  threads(nthre);
		for(int i=0; i<nloop; i++){
			kernel<<< grid, threads >>>(dval, nword);
			//		cutilSafeCall( hipDeviceSynchronize() );  // no need !
		}
		cutilCheckMsg("Kernel execution failed");


		cutilSafeCall( hipMemcpy(hval, dval, mem_size, hipMemcpyDeviceToHost) );
		tt = t.get_time() - tt;

		int nerr=0;
		for(int i=0; i<nword; i++){
			int x = hval[i];
			int z = zval[i];
			if(x != z) {
				nerr++;
#if defined(DEBUG)
				if(nerr<5) printf("%08d(%03d MB): %08x %08x %08x\n", i, i*sizeof(int)/1024/1024, x, z, x^z);
#endif
			}
		}

		if(nerr > 0){
			printf("FAILED[%d]: %d errors, %f\n", gpuid, nerr, tt);
		}else{
			printf("SUCCESS[%d]: %d errors, %f\n", gpuid, nerr, tt);
		}

		cutilSafeCall(hipFree(dval));
		free(hval);
		free(zval);
	}
};

int main( int argc, char** argv) 
{
	int nloop = 10240;
	GPU gpu[4];
	int tid[4]={-1, -1, -1, -1};

#pragma omp parallel for
	for(int i=0; i<4 ; i++){
		tid[i] = omp_get_thread_num(); // check
		gpu[i].open(i);
		int devid = gpu[i].GetDevice(); // check
		printf("tid %d, devid %d\n", tid[i], devid);
		gpu[i].runtest(i, nloop*2);
	}
}



