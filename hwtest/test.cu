#include "hip/hip_runtime.h"
//Time-stamp: <2011-01-02 06:32:30 hamada>
//#define DEBUG

#include <iostream>
#include <string.h>
#include <cassert>
#include <cutil.h>

//--------------------------------
#include <sys/time.h>
#include <sys/resource.h>

class Time{
	double t;
public:
	Time() {t = 0.0;}
	struct timeval tv;
	struct timezone tz;
	double get_time(void){
		gettimeofday(&tv, &tz);
		t = (double)(tv.tv_sec  + tv.tv_usec*1.0e-6);
		return (t);
	}
};

__global__ void kernel(int* dval, int nword)
{
  int tid = threadIdx.x;
	int3 bid;
	bid.x = blockIdx.x;
	bid.y = blockIdx.y;

	int nthre = blockDim.x;
	int i   = (gridDim.x*bid.y + bid.x)*nthre + tid;

	dval[i] = ~dval[i];
}

#include <ctime>

void setup_vector(int n, int *x0,  int *x1)
{
	//	srand48(0x19740526);
	srand48((long int)time(NULL));
	for(int i=0; i<n; i++){
		x0[i]= (int)(0xFFFFFFFFull & mrand48());
		x1[i]=x0[i];
	}

	/*
	int nn=0;
	for(int i=0; i<n; i++) if(x0[i]<0) nn++;
	printf("%d, %d, %1.16f\n", n, nn, (double)nn/(double)n);
	*/
}

#include <cutil_inline.h>

int main( int argc, char** argv) 
{
	assert(argc==3);
	int gpuid = atoi(argv[1]);
	int nloop = atoi(argv[2]) * 2; // 10240 : 3 min
	assert(gpuid<4);
	assert(gpuid>=0);
	assert(nloop>0);

#if defined(DEBUG)
	printf("gpuid = %d\n", gpuid);
	printf("nloop = %d\n", nloop);
#endif

	cutilSafeCall( hipSetDevice(gpuid) );

	int nby   = 6;
	int nbx   = 65535; // max 65535 blocks

	int nthre = 512;   // max 512 threads
	int nword = nbx * nby * nthre;
	int mem_size = sizeof(int) * nword;

#if defined(DEBUG)
	printf("# threads:   %d \n", nword);
	printf("mem_size:    %d Kbyte\n", mem_size >> 10);
#endif

	int* hval = (int*) malloc(mem_size);
	int* zval = (int*) malloc(mem_size);
	int* dval;
	cutilSafeCall( hipMalloc( (void**) &dval, mem_size) );

	setup_vector(nword, hval, zval);

	Time t;
	double tt;
	tt = t.get_time();

	cutilSafeCall( hipMemcpy(dval, hval, mem_size, hipMemcpyHostToDevice) );
	dim3  grid(nbx, nby);
	dim3  threads(nthre);

	for(int i=0; i<nloop; i++){
	  kernel<<< grid, threads >>>(dval, nword);
		//		cutilSafeCall( hipDeviceSynchronize() );  // no need !
	}
	cutilSafeCall( hipMemcpy(hval, dval, mem_size, hipMemcpyDeviceToHost) );

	tt = t.get_time() - tt;

	int nerr=0;
	for(int i=0; i<nword; i++){
		int x = hval[i];
		int z = zval[i];
		if(x != z) {
		  nerr++;
#if defined(DEBUG)
		  if(nerr<5) printf("%08d(%03d MB): %08x %08x %08x\n", i, i*sizeof(int)/1024/1024, x, z, x^z);
#endif
		}
	}

	if(nerr > 0){
		printf("FAILED[%d]: %d errors, %f\n", gpuid, nerr, tt);
	}else{
		printf("SUCCESS[%d]: %d errors, %f\n", gpuid, nerr, tt);
	}

	cutilSafeCall(hipFree(dval));
	free(hval);
	free(zval);
	//	CUDA_SAFE_CALL(hipFree(dval));
	return (0);
}
