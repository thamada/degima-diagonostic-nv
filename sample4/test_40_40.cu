
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <math.h>
using namespace std;

namespace myNamespace_40_40{
	static double* hmem_i;
	static double* hmem_o;
	static double* dmem_i;
	static double* dmem_o;
	static hipStream_t stream;
	static int nb    = 1;   //1024*1024*64*2; // max 1024*1024*64*2
	static int nthre = 1; // max 65535
	static int nthre_total = nb*nthre;
	static int nword =  1024;
	static int mem_size = sizeof(double) * nword;
	static int mem_size_o = nthre_total*sizeof(double);

	__device__ double myDeviceFunc(double* in, int nword)
	{
		double z=0.0;

		for(int i=0; i<nword; i++)
			z += in[i];

		return (z);
	} 

	__global__ void kernel(double* in, double* out, int nword)
	{
		int tid = threadIdx.x;
		int bid = blockIdx.x;
		int index = blockDim.x*bid + tid;
		double z = myDeviceFunc(in, nword);
		out[index] = z;
	}

	void initialize()
	{
		static bool is_first = true;
		if(false == is_first) return;

		// setup stream
		hipStreamCreate(&stream);
		// input buffer (Host)
		hmem_i = (double*) malloc(mem_size);
		for(int i=0; i<nword; i++) hmem_i[i] = (double)i;
		// input buffer (GPU)
		hipMalloc( (void**) &dmem_i, mem_size);
		hipMemcpyAsync(dmem_i, hmem_i, mem_size, hipMemcpyHostToDevice, stream);
		// output buffer (Host/GPU)
		hipMalloc( (void**) &dmem_o, mem_size_o);
		hmem_o = (double*) malloc(mem_size_o);

		printf("stream #: %d\n", stream);
		printf("# threads:   %d \n", nthre_total);
		printf("mem_size:    %d MB\n", mem_size >> 20);
		printf("mem_size_o:    %d kB\n", mem_size_o >> 10);
		is_first = false;
	}

	void run(int n_run)
	{
		kernel<<< nb, nthre, 0, stream >>>(dmem_i, dmem_o, nword);
		hipMemcpyAsync(hmem_o, dmem_o, mem_size_o, hipMemcpyDeviceToHost, stream);

		/*
		for(int i=0; i<nthre_total; i++){
			double z = hmem_o[i];
			if(i>(nthre_total-4)) printf("%d, %f\n", i, z);
		}
		*/

		printf("%d: %d, %f\n", stream, nthre_total-1, hmem_o[nthre_total-1]);

		//		if(n_run % 32 == 31) cudaStreamSynchronize(stream);

		return;
	}

	void finalize(){
		free(hmem_i);
		free(hmem_o);
		hipFree(dmem_i);
		hipFree(dmem_o);
		hipStreamSynchronize(stream);
		hipStreamDestroy(stream);
	}
}

