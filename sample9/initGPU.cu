
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <assert.h>
using namespace std;

extern "C"
int initGPU(int mpi_proc_id)
{
	hipError_t err;

	// total number of GPUs
	int n_gpu;
	err = hipGetDeviceCount(&n_gpu);
	assert(err == hipSuccess);
	assert(n_gpu > 0);

	// select a GPU
	int devid = mpi_proc_id % n_gpu;
	err = hipSetDevice(devid);
	assert(err == hipSuccess);

	// check device Id
	int devid2 = -1;
	err = hipGetDevice(&devid2);
	assert(err == hipSuccess);
	assert(devid == devid2);	

	return (devid);
}
